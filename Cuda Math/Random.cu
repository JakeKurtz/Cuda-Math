#include "hip/hip_runtime.h"
/* ---------------------------------------------------------------------------------- *
*
*    MIT License
*
*    Copyright(c) 2024 Jake Kurtz
*
*    Permission is hereby granted, free of charge, to any person obtaining a copy
*    of this softwareand associated documentation files(the "Software"), to deal
*    in the Software without restriction, including without limitation the rights
*    to use, copy, modify, merge, publish, distribute, sublicense, and /or sell
*    copies of the Software, and to permit persons to whom the Software is
*    furnished to do so, subject to the following conditions :
*
*    The above copyright noticeand this permission notice shall be included in all
*    copies or substantial portions of the Software.
*
*    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
*    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
*    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
*    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
*    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
*    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
*    SOFTWARE.
*
* ---------------------------------------------------------------------------------- */

#include "random.h"

namespace cml
{
	uint32_t lowerbias32(uint32_t x)
	{
		x ^= x >> 16;
		x *= 0xa812d533;
		x ^= x >> 15;
		x *= 0xb278e4ad;
		x ^= x >> 17;
		return x;
	}

	// Note:	host rand range:	[0, 32767)
	//			device rand range:	[0, 4294967296)

	uint32_t rand()
	{
#if defined(__CUDA_ARCH__)
		uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;

		uint32_t seed = (i * 256 + j);
		return lowerbias32(seed);
#else
		return ::rand();
#endif
	}

	float rand_float()
	{
#if defined(__CUDA_ARCH__)
		return rand() * 0.00000000023283064365386962890625;
#else
		return rand() * 0.00003051850947599719229712820825;
#endif
	}
	float rand_float(float min, float max)
	{
		return ((rand_float() * (max - min)) + min);
	}

	int rand_int()
	{
		return rand();
	}
	int rand_int(int min, int max)
	{
		return ((rand_float() * (max - min)) + min);
	}
}